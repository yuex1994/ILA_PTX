
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
__global__ void add_neighbour(int *A, int offset) { 
      int tid = threadIdx.x; 
        A[tid] += A[tid + offset]; 
}

const int N = 8192;
const int threadsPerBlock = 128;
const int blockPerGrid = 64;
int main(){
    int* A, *devA;
    A = new int[N];
    hipMalloc((void **) &devA, sizeof(int) * N);
    for (int i = 0; i < N; i++)
        A[i] = i;
    hipMemcpy(devA, A, N * sizeof(int), hipMemcpyHostToDevice);
    add_neighbour<<<blockPerGrid, threadsPerBlock>>>(devA, 1);
    hipMemcpy(A, devA, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0 ; i < N; i++)
        printf("%d \n", A[i]); 
    return 1;
}
