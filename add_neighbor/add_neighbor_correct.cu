
#include <hip/hip_runtime.h>
//--blockDim=1024 --gridDim=1

/* 
 * The intention of this kernel is to increment each
  * element of 'A' with its neighbouring element,
   * 'offset' places away.
    *
     * A barrier statement ensures that read-write data
      * races do not occur.
       */


       __global__ void add_neighbour(int *A, int offset) { 
             unsigned tid = threadIdx.x; 

               // use a barrier to order the accesses to A
                 int temp = A[tid + offset];
                   __syncthreads();
                     A[tid] += temp;
       }


int main() {
    return 0;
}
