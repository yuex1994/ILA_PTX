
#include <hip/hip_runtime.h>
__global__ void two_threads(int *A, int *B) {
    int tid = threadIdx.x;
    A[tid] += B[tid];
}
