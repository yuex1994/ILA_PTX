#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define imin(a,b) (a<b?a:b)
#define BAR_INIT 0
#define BAR_ENTER 1
#define BAR_WAIT 2
#define BAR_EXIT 3
#define BAR_FINISH 4

const int N = 128;
const int threadsPerBlock = 128;
const int blocksPerGrid = 1;
/*
__global__ void dot(int *a, int *b, int *c, int *d) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x; 
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
    __shared__ int counter_enter;
    __shared__ int counter_exit;
    counter_enter = 0;
    counter_exit = N;
    //__syncthreads();
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid == 0) {
    
    } else {
    }
    //atomic add 1 to counter
    int r = atomicAdd(&counter_enter, 1); 
    for (;;) {
        if (r == N)
            break;
        else
            r = atomicAdd(&counter_enter, 0);
    }
    c[gid] = r;
    r = atomicSub(&counter_exit, 1);
    //__syncthreads();
    while(r != 0) {
        r = atomicSub(&counter_exit, 0);
    }
    d[gid] = r;
    //d[gid] = counter_exit;
    //printf("post_print, %d\n", gid);
//    if (gid % 2 == 1)
//    else
//        c[gid] =  1;
//    c[gid] = c[gid] + 1;
}*/
//__global__ void dot2(float *a, float *b, float*c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x;
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
//}
//__global__ void mykernel(int *data){
//    atomicAdd(data, 10); 
//}
__global__ void dot(int *bar_state_array, int *bar_counter_enter_array, int *bar_counter_exit_array) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    int bar_state = bar_state_array[gid];
    int bar_counter_enter = bar_counter_enter_array[gid];
    int bar_counter_exit = bar_counter_exit_array[gid];
    if (gid == 0) {
        if (bar_state == BAR_INIT) {
            bar_state = BAR_ENTER;
        } else if(bar_state == BAR_ENTER) {
            if (bar_counter_exit == 0) {
                if (bar_counter_enter == (N-1)) {
                    bar_counter_enter = N;
                    bar_state = BAR_EXIT;
                    bar_counter_exit = N;
                }
                else {
                    bar_counter_enter += 1;
                    bar_state = BAR_WAIT;
                }
            }
        } else if(bar_state == BAR_WAIT) {
            if (bar_counter_enter == N) {
                bar_state = BAR_EXIT;
            }
        } else if (bar_state == BAR_EXIT) {
            if (bar_counter_exit == 1) {
                bar_counter_enter = 0;
                bar_counter_exit = 0;
                bar_state = BAR_FINISH;
            } else {
                bar_counter_exit -= 1;
                bar_state = BAR_FINISH;
            }
        } else if (bar_state == BAR_FINISH) 
            bar_state = BAR_INIT;
    }
    bar_state_array[gid] = bar_state;
    bar_counter_enter_array[gid] = bar_counter_enter;
    bar_counter_exit_array[gid] = bar_counter_exit;


}
int main(){
    //int *a, *b, *partial_c, *partial_d;
    int *bar_state_array, *bar_counter_enter_array, *bar_counter_exit_array;
    int *dev_bar_state_array, *dev_bar_counter_enter_array, *dev_bar_counter_exit_array; 
    
    bar_state_array = new int[N];
    bar_counter_enter_array = new int[N];
    bar_counter_exit_array = new int[N];

    hipMalloc((void **)&dev_bar_state_array, sizeof(int) * N);
    hipMalloc((void **)&dev_bar_counter_enter_array, sizeof(int) * N);
    hipMalloc((void **)&dev_bar_counter_exit_array, sizeof(int) * N);
    for (int i = 0; i < N; i++) {
        bar_state_array[i] = 0;
        bar_counter_enter_array[i] = 0;
        bar_counter_exit_array[i] = 0;
    }
    bar_state_array[0] = 1;
    bar_counter_enter_array[0] = N - 1;
    bar_counter_exit_array[0] = 0;
    hipMemcpy(dev_bar_state_array, bar_state_array, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_bar_counter_enter_array, bar_counter_enter_array, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_bar_counter_exit_array, bar_counter_exit_array, N*sizeof(int), hipMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_bar_state_array, dev_bar_counter_enter_array, dev_bar_counter_exit_array);
    hipMemcpy(bar_state_array, dev_bar_state_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bar_counter_enter_array, dev_bar_counter_enter_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(bar_counter_exit_array, dev_bar_counter_exit_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_bar_state_array);
    hipFree(dev_bar_counter_enter_array);
    hipFree(dev_bar_counter_exit_array);
    printf("%d\n", bar_state_array[0]);
    printf("%d\n", bar_counter_enter_array[0]);
    printf("%d\n", bar_counter_exit_array[0]);
    delete[] bar_state_array;
    delete[] bar_counter_enter_array;
    delete[] bar_counter_exit_array;
    //int *dev_a, *dev_b, *dev_partial_c, *dev_partial_d;
    /*
    a = new int[N];
    b = new int[N];
    partial_c = new int[N];
    partial_d = new int[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2.0f;
        partial_c[i] = 200;
        partial_d[i] = 250;
    }
    cudaMalloc((void **)&dev_a, sizeof(int) * N);
    cudaMalloc((void **)&dev_b, sizeof(int) * N);
    cudaMalloc((void **)&dev_partial_c, N*sizeof(int));
    cudaMalloc((void **)&dev_partial_d, N*sizeof(int));
    cudaMemcpy(dev_a, a, N*sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(dev_b, b, N*sizeof(int), cudaMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c, dev_partial_d);
    cudaMemcpy( partial_c, dev_partial_c, N*sizeof(int), cudaMemcpyDeviceToHost);
    cudaMemcpy( partial_d, dev_partial_d, N*sizeof(int), cudaMemcpyDeviceToHost);
    #define sum_sq(x) (x*(x+1)*(2*x+1)/6)
    
    cudaFree(dev_a);
    cudaFree(dev_b);
    cudaFree(dev_partial_c);
    cudaFree(dev_partial_d);
    for(int i = 0 ; i < N; i++) {
        printf("enter %d has %d\n", i, partial_c[i]);
    }
    for (int i = 0; i < N; i++) {
        printf("exit %d, has %d\n", i, partial_d[i]);
    }
    
    delete[]  a;
    delete[]  b;
    delete[]  partial_c;
    */
}
