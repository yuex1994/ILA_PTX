
#include <hip/hip_runtime.h>
__global__ void f(int * A, int *B) {

    int tid = threadIdx.x;
    int diff = (B - A);

    int x = B[tid];
    int y = A[tid + diff - 1];

    B[tid] = x + y;

}

