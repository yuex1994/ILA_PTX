
#include <hip/hip_runtime.h>
//--blockDim=1024 --gridDim=1

/*
 * A single thread block collaborate to perform
  * a sum reduction on the array of floats 'in'.
   * The reduction result is written to the
    * address 'result'.  The number of elements to
     * be reduced is given by 'size'
      *
       * This example may take a while to verify as it
        * requires non-trivial loop invariants to be
         * inferred.
          */

#define N 8192 * 1024 /* Same as blockDim */

#define tid threadIdx.x

__global__ void reduce(float *in, float *result, unsigned size) {

      __shared__ float partial_sums[N];

        /* Each thread sums elements
             in[tid], in[tid + N], in[tid + 2*N], ...
               */
                 partial_sums[tid] = in[tid];
                   for(int i = tid + N; i < size; i += N) {
                           partial_sums[i] += in[i];
                             }

                               __syncthreads();

                                 /* Tree reduction computes final sum into partial_sums[0] */
                                   for(int d = N/2; d > 0; d >>= 1) {
                                           if(tid < d) {
                                                     partial_sums[tid] += partial_sums[tid + d];
                                                         }
                                                             __syncthreads();
                                                               }

                                                                 /* Master thread writes out result */
                                                                   if(tid == 0) {
                                                                           *result = partial_sums[0];
                                                                             }
                                                                               
}

