#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define imin(a,b) (a<b?a:b)

const int N = 4;
const int threadsPerBlock = 4;
const int blocksPerGrid = 1;

__global__ void dot(int *a) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x; 
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
    int tid = threadIdx.x;
    if ((tid % 2) == 1){
        int t0 = a[tid - 1];
        int t1 = a[tid];
        a[tid] = t0+t1;

    } else {
        int t0 = a[tid + 1];
        int t1 = a[tid];
        a[tid] = t0 + t1;
 
    }
//    else
//        c[gid] =  1;
//    c[gid] = c[gid] + 1;
}
//__global__ void dot2(int *a, int *b, int*c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x;
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
//}
//__global__ void mykernel(int *data){
//    atomicAdd(data, 10); 
//}
   
int main(){
    int *a;
    int *dev_a;
    
    a = new int[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
    }
    hipMalloc((void **)&dev_a, sizeof(int) * N);
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a);
    hipMemcpy( a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);
    #define sum_sq(x) (x*(x+1)*(2*x+1)/6)

    for (int i = 0;i < N; i++)
        printf("%d\n", a[i]);
    hipFree(dev_a);
    

    delete[]  a;
}
