
#include <hip/hip_runtime.h>
__device__ int test(int tid) {
    tid += 1;
    __syncthreads();
    return tid;
}

__global__ void tfunction(int* A) {
    int tid = threadIdx.x;
    A[tid] = test(tid);
    
}
