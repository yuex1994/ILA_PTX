
#include <hip/hip_runtime.h>
__global__ void simple_loop(int *a, int size) {
    int tid = threadIdx.x;
    for (int i = 0; i < size; i++) {
        a[tid * 3] += a[tid * 3 + i];
    }
}
