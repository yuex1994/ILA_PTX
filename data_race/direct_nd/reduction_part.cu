
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
        /* a sum reduction on the array of floats 'in'.
         * The reduction result is written to the
          * address 'result'.  The number of elements to
           * be reduced is given by 'size'
            *
             * The example contains data races because barrier
              * synchronisation statements, of the form:
               *   __syncthreads();
                * are missing.
                 *
                  * Can you add them to eliminate all data races?
                   */

#define N 4/* Same as blockDim */


__global__ void reduce(int *in) {
    int tid = threadIdx.x;
    for(int d = N/2; d > 0; d >>= 1) {
                                         if(tid < d) {
                                                   in[tid] += in[tid + d];
                                                       }
                                                       __syncthreads();
                                                         }

                                                           /* Master thread writes out result */
                                                                         
}

