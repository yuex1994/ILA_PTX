
#include <hip/hip_runtime.h>
__global__ void simple_loop(int *a) {
    int tid = threadIdx.x;
    for (int i = 0; i < 5; i++) {
        a[tid * 4] += a[tid * 4 + i];
    }
}
