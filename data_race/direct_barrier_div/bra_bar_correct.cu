
#include <hip/hip_runtime.h>
__global__ void branch(int *A){
    int tid = threadIdx.x;
    if ((tid % 2) == 1) {
        int a = A[tid + 2];
        __syncthreads();
        A[tid] += a;
    }
}
