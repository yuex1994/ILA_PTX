
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;
__global__ void add_neighbour(int *A) { 
      int tid = threadIdx.x; 
/*      int a = A[tid + offset];
      printf("%d, %d\n", tid, a);
      __syncthreads();
      A[tid + offset] = A[tid]; 
      __syncthreads();
      A[tid] = A[tid + offset];
      printf("%d, %d\n",tid ,A[tid]);*/
      A[tid] += A[tid + 2];//A[tid + offset];
}

const int N = 2;
const int threadsPerBlock = 2;
const int blockPerGrid = 1;
int main(){
    int* A, *devA;
    A = new int[N];
    hipMalloc((void **) &devA, sizeof(int) * N);
    for (int i = 0; i < N; i++)
        A[i] = i;
    hipMemcpy(devA, A, N * sizeof(int), hipMemcpyHostToDevice);
    add_neighbour<<<blockPerGrid, threadsPerBlock>>>(devA);
    hipMemcpy(A, devA, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0 ; i < N; i++)
        printf("%d \n", A[i]); 
    return 1;
}
