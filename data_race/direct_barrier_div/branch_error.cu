
#include <hip/hip_runtime.h>
__global__ void branch(int *A) {
    int tid = threadIdx.x;
    if ((tid % 2) == 1) {
        A[tid] += A[tid + 2];
    }
}
