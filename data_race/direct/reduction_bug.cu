#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
using namespace std;
        /* a sum reduction on the array of floats 'in'.
         * The reduction result is written to the
          * address 'result'.  The number of elements to
           * be reduced is given by 'size'
            *
             * The example contains data races because barrier
              * synchronisation statements, of the form:
               *   __syncthreads();
                * are missing.
                 *
                  * Can you add them to eliminate all data races?
                   */

#define N 2 /* Same as blockDim */

#define tid threadIdx.x

__global__ void reduce(int *in, int *result, int size) {

      __shared__ int partial_sums[N];

        /* Each thread sums elements
             in[tid], in[tid + N], in[tid + 2*N], ...
               */
                 partial_sums[tid] = in[tid];
                   for(int i = tid + N; i < size; i += N) {
                           partial_sums[i] += in[i];
                             }

                               /* Tree reduction computes final sum into partial_sums[0] */
                                 for(int d = N/2; d > 0; d >>= 1) {
                                         if(tid < d) {
                                                   partial_sums[tid] += partial_sums[tid + d];
                                                       }
                                                         }

                                                           /* Master thread writes out result */
                                                             if(tid == 0) {
                                                                     *result = partial_sums[0];
                                                                       }
                                                                         
}

const int threadsPerBlock = N;
int main(){
    int* A, *devA, *B, *devB;
    int size = 8192;
    A = new int[N];
    B = new int;
    hipMalloc((void **) &devA, sizeof(int) * N);
    hipMalloc((void **) &devB, sizeof(int));
    cudaMallov
    for (int i = 0; i < N; i++)
        A[i] = i;
    hipMemcpy(devA, A, N * sizeof(int), hipMemcpyHostToDevice);
    reduce<<<1, threadsPerBlock>>>(devA, devB);
    hipMemcpy(A, devA, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(B, devB, N * sizeof(int), hipMemcpyDeviceToHost);
    return 1;
}
