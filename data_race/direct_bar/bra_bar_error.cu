
#include <hip/hip_runtime.h>
__global__ void branch(int *A){
    int tid = threadIdx.x;
    if ((tid % 2) == 1) {
        __syncthreads();
        int a = A[tid + 2];
        A[tid] += a;
    }
}
